
#include <hip/hip_runtime.h>
__global__ 
void Mandelbrot(float* out,const double* re,const double* im){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    double zr=0;
    double zi=0;
    double cr=re[idx];
    double ci=im[idx];
    double count=0;
    for(int i=1;i<1000;i++){
        double zrt=zr*zr-zi*zi;
        double zit=2*zr*zi;
        zr=zrt+cr;
        zi=zit+ci;
        if(zr*zr+zi*zi<4)count++;
    }
    out[idx]=count;
}